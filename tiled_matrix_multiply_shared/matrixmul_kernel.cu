#include "hip/hip_runtime.h"
/* Matrix multiplication: P = M * N.
 * Device code.

    Author: Naga Kandasamy
    Date: 2/16/2017
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_
#define TILE_SIZE 32
#include <stdio.h>
#include "matrixmul.h"

__global__ void 
MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{

	__shared__ float Msub[TILE_SIZE][TILE_SIZE];
	__shared__ float Nsub[TILE_SIZE];
	// Thread index
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;

	// Block index
	int blockX = blockIdx.y;
	//int blockY = blockIdx.y;

	// Find position in Matrix
	//int column_number = blockDim.x * blockX + threadX;
	int row_number = ((TILE_SIZE)* blockX) + threadY;
	

	double P_temp = 0.0f;

	//this is where things get funky

	int k = 0;
	int temp;
	while(k < M.width){

        	if(k + threadX  < M.width && row_number < M.height)
			Msub[threadY][threadX] = M.elements[row_number*M.width + k + threadX];
		else
			Msub[threadY][threadX] = 0.0f;
		
		if(k + threadY < N.height)
			Nsub[threadX] = N.elements[threadX + k];

		else
			Nsub[threadY] = 0.0f;
		for(temp = 0;temp <TILE_SIZE; temp++)
			P_temp += Msub[threadY][temp]*Nsub[temp];
		__syncthreads();
		k += TILE_SIZE;
	}		
	
		
	// Write result to P
	//P[row_number * matrix_size + column_number] = (float)P_temp;
	if(row_number < P.height)
		P.elements[row_number] = (float)P_temp;
	return;
}

#endif
