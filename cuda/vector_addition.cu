//LOOK AT ONE OF THE INCLUDE FILES. DOES NOT SEEM RIGHT
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <float.h>
#include <string,h>

#include "vector_addition_kernel.cu"

#define THREAD_BLOCK_SIZE 128
#define NUM_THREAD_BLOCKS 240



void compute_on_device(float *A_on_host,float *B_on_host, float *gpu_result,int num_elements)
{

float *A_on_device = NULL;
float *B_on_device = NULL;
float *C_on_device = NULL;




//create  a space in GPU for A
hipMalloc((void **) &A_on_device, num_elements*sizeof(float));

//copy elements from CPU to GPU
hipMemcpy(A_on_device,A_on_host, num_elements*sizeof(float), hipMemcpyHostToDevice); //copy from host to device a set number of elements




//create  a space in GPU for A
hipMalloc((void **) &B_on_device, num_elements*sizeof(float));I
hipMemcpy(B_on_device,B_on_host, num_elements*sizeof(float), hipMemcpyHostToDevice); //copy from host to device a set number of elements

//llocate space in memory on GPU for the output
hipMalloc((void **) &C_on_device, num_elements*sizeof(float));

//setup execution grid on GPU
dim3 thread_block(THREAD_BLOCK_SIZE,1,1); //set number of threads in a thread block
printf("Setting a (%d x 1) execution grid .\n",NUM_THREAD_BLOCKS);
dim3 grid(NUM_THREAD_BLOCKS, 1);

printf("Adding vectors on the GPU. \n");





