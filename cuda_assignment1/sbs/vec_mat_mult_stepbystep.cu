#include "hip/hip_runtime.h"
/* Vector matrix multiplication
* Host Code
* Date: 02/26/2018
*/

//include all required packages
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include "vec_mat_mult.h" //this contains the structure
#include "matrixmul_kernel.cu"
//will need to input kernel into this

#define MATRIX_SIZE 512 //would like this to be a user input eventually
#define TILE_SIZE 32

extern "C" void compute_gold(float *,const float*,const float*, unsigned int, unsigned int); //this is an extern function that I am importing
Matrix AllocateMatrix(int height, int width, int init);
void fdm(Matrix* M);

void MatrixMulOnDevice(const Matrix M, const Matrix N, Matrix P);


int main(int argc, char **argv){

	Matrix M,N,P;


	srand(52);
	M = AllocateMatrix(MATRIX_SIZE,MATRIX_SIZE,1);
	N = AllocateMatrix(MATRIX_SIZE,1,1);
	P = AllocateMatrix(MATRIX_SIZE,1,0);

	printf(" multiplying matrices on a GPU \n");

	MatrixMulOnDevice(M,N,P);
	

	printf("Multiplying serially on CPU \n");
//this is to cmom[pute in sequential
	struct timeval start, stop;
	gettimeofday(&start, NULL);
	Matrix reference = AllocateMatrix(P.num_rows,P.num_columns,0);
	compute_gold(reference.elements,M.elements,N.elements,M.num_rows,N.num_columns);
	

	gettimeofday(&stop, NULL);
	printf("Execution time = %fs. \n", (float)(stop.tv_sec - start.tv_sec+ (stop.tv_usec - start.tv_usec)/(float)100000));

}

void fdm(Matrix* M){
	hipFree(M->elements);
	M->elements = NULL;

}

//FUNCTION CALLED ALLOCATE MATRIX
Matrix
AllocateMatrix(int height, int width, int init)
{
	Matrix M;
	M.num_columns = width;
	M.num_rows  = height;
	int size = M.num_columns*M.num_rows;
	M.elements = (float*) malloc(size*sizeof(float));



	for(unsigned int i = 0; i < M.num_rows * M.num_columns ;i++){

		M.elements[i] = (init == 0) ? (0.0f) : floor((rand()*3 / (float)RAND_MAX));
	}



	return M;

}

void
MatrixMulOnDevice(const Matrix M, const Matrix N, Matrix P){
	Matrix M_on_device;
	Matrix N_on_device;
	Matrix P_on_device;


//this line is used to allocate memory
	hipMalloc((void **) &M_on_device.elements, M.num_columns*M.num_rows*sizeof(float));
	hipMemcpy(M_on_device.elements,M.elements,M.num_columns*M.num_rows*sizeof(float),hipMemcpyHostToDevice);

	hipMalloc((void **) &N_on_device.elements, N.num_columns*N.num_rows*sizeof(float));
	hipMemcpy(N_on_device.elements,N.elements,N.num_columns*N.num_rows*sizeof(float),hipMemcpyHostToDevice);


	hipMalloc((void **) &P_on_device.elements, M.num_rows*N.num_columns*sizeof(float));

	dim3 threads(TILE_SIZE,TILE_SIZE); //intitalize a  thread warp


//determine the size of the execution gtid
	dim3 grid((P_on_device.num_columns + TILE_SIZE - 1)/TILE_SIZE, (P_on_device.num_rows + TILE_SIZE - 1)/TILE_SIZE);


	struct timeval start, stop;
	gettimeofday(&start, NULL);

	MatrixMulKernel<<< grid, threads >>> (P_on_device.elements, M_on_device.elements, N_on_device.elements, MATRIX_SIZE);
	
	hipDeviceSynchronize();




	gettimeofday(&stop, NULL);
	//printf("Execution time = %fs. \n" (float) (stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));


//print to host
	int size = P.num_columns*P.num_rows*sizeof(float);
	hipMemcpy(P.elements,P_on_device.elements,size,hipMemcpyDeviceToHost);

//free the matrix
	fdm(&M_on_device);
	fdm(&N_on_device);
	fdm(&P_on_device);




}

















	


