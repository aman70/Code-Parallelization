
#include <hip/hip_runtime.h>
//kernel fior tiled multiplication


__global__ void
MatrixMulKernel(float* P, const float* M, const float* N, int matrix_size){
	//thread index
//	int threadX = threadIdx.x;
	int threadY = threadIdx.y;


	//block index
//	int blockX = blockIdx.x;
	int blockY = blockIdx.y;


	//FIND position in gloval matrix

	//int column_number = blockDim.x*blockX + threadX;

	int row_number = blockDim.y*blockY + threadY;
	

	double P_temp = 0;

	int k;

	for( k = 0;k< matrix_size; k++){
		double M_element = M[matrix_size*row_number + k];
		double N_element = N[k];
		P_temp += M_element*N_element;

	}	
	P[k] = (float)P_temp;

}
